/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

// TODO: Clean up includes when all debugging is done.
#include <cudf/column/column.hpp>
#include <cudf/column/column_factories.hpp>
#include <memory>
#include <cudf/binning/bin.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/types.hpp>
#include <thrust/functional.h>
#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/tuple.h>
#include <stdio.h>
#include <vector>
#include <numeric>
#include <cudf/copying.hpp>

namespace cudf {

namespace bin {

constexpr unsigned int MYNULL = 0xffffffff;

template <typename StrictWeakOrderingLeft, typename StrictWeakOrderingRight>
struct bin_finder
{
    bin_finder(
            const float *left_edges,
            const float *left_edges_end,
            const float *right_edges
            )
        : m_left_edges(left_edges), m_left_edges_end(left_edges_end), m_right_edges(right_edges),
          m_left_comp(StrictWeakOrderingLeft()), m_right_comp(StrictWeakOrderingRight())
    {}

    __device__ unsigned int operator()(const float value) const
    {
        // TODO: Immediately return NULL for NULL values.
        auto bound = thrust::lower_bound(thrust::seq,
                m_left_edges, m_left_edges_end,
                value,
                m_left_comp);

        // First check if the input is actually contained in the interval; if not, assign MYNULL.
        if ((bound == m_left_edges) || (bound == m_left_edges_end))
            return MYNULL;

        // We must subtract 1 because lower bound returns the first index _greater than_ the value.
        auto index = bound - m_left_edges - 1;
        return (m_right_comp(value, m_right_edges[index])) ? index : MYNULL;
    }

    const float *m_left_edges;
    const float *m_left_edges_end;
    const float *m_right_edges;
    // TODO: Can I store these by reference? Don't think so since the argument
    // to lower_bound is not a ref, but I should check to be sure.
    StrictWeakOrderingLeft m_left_comp;
    StrictWeakOrderingRight m_right_comp;
};


// Bin the input by the edges in left_edges and right_edges.
template <typename StrictWeakOrderingLeft, typename StrictWeakOrderingRight>
std::unique_ptr<column> bin_internal(column_view const& input, 
                            column_view const& left_edges,
                            column_view const& right_edges,
                            rmm::mr::device_memory_resource * mr)
{
    // TODO: Add check that edge sizes are > 0.
    CUDF_EXPECTS(input.type() == left_edges.type(), "The input and edge columns must have the same types.");
    CUDF_EXPECTS(input.type() == right_edges.type(), "The input and edge columns must have the same types.");
    CUDF_EXPECTS(left_edges.size() == right_edges.size(), "The left and right edge columns must be of the same length.");

    // Handle empty inputs.
    if (input.is_empty()) {
        // TODO: Determine what output type actually makes sense here, it
        // probably shouldn't be empty_like but instead of some numeric type.
        return empty_like(input);
    }

    // TODO: Figure out how to get these two template type from the input.
    auto output = cudf::make_numeric_column(input.type(), input.size());

    thrust::transform(thrust::device,
            input.begin<float>(), input.end<float>(),
            static_cast<cudf::mutable_column_view>(*output).begin<unsigned int>(),
            bin_finder<StrictWeakOrderingLeft, StrictWeakOrderingRight>(
                left_edges.begin<float>(), left_edges.end<float>(), right_edges.begin<float>()
                )
            );

    //unsigned int *tmp = (unsigned int *) malloc(10 * sizeof(unsigned int));
    //hipError_t err = hipMemcpy(tmp, static_cast<cudf::mutable_column_view>(*output).begin<unsigned int>(), 10 * sizeof(unsigned int), hipMemcpyDeviceToHost);
    //fprintf(stderr, "The values of the output are %d, %d, %d.\n", tmp[0], tmp[1], tmp[2]);

    return output;
}



template <typename T>
constexpr inline auto is_supported_bin_type()
{
  return (cudf::is_numeric<T>() && not std::is_same<T, bool>::value); // || cudf::is_fixed_point<T>();
}


struct bin_type_dispatcher {
    template <typename T, typename... Args>
    std::enable_if_t<not is_supported_bin_type<T>(), std::unique_ptr<column>> operator()(
            Args&&... args)
    {
        CUDF_FAIL("Type not support for cudf::bin");
    }

    template <typename T>
    std::enable_if_t<is_supported_bin_type<T>(), std::unique_ptr<column>> operator()(
            column_view const& input, 
            column_view const& left_edges,
            inclusive left_inclusive,
            column_view const& right_edges,
            inclusive right_inclusive,
            rmm::mr::device_memory_resource * mr)
    {
        switch (left_inclusive)
        {
            case inclusive::YES:
                switch (right_inclusive)
                {
                    case inclusive::YES:
                        // TODO: Don't pick float by default, dispatch as needed.
                        return bin_internal<thrust::less_equal<float>, thrust::less_equal<float> >(input, left_edges, right_edges, mr);
                    case inclusive::NO:
                        return bin_internal<thrust::less_equal<float>, thrust::less<float> >(input, left_edges, right_edges, mr);
                }
            case inclusive::NO:
                switch (right_inclusive)
                {
                    case inclusive::YES:
                        return bin_internal<thrust::less<float>, thrust::less_equal<float> >(input, left_edges, right_edges, mr);
                    case inclusive::NO:
                        return bin_internal<thrust::less<float>, thrust::less<float> >(input, left_edges, right_edges, mr);
                }
            default:
                CUDF_FAIL("Undefined rounding method");
        }
    }
};



// Bin the input by the edges in left_edges and right_edges.
std::unique_ptr<column> bin(column_view const& input, 
                            column_view const& left_edges,
                            inclusive left_inclusive,
                            column_view const& right_edges,
                            inclusive right_inclusive,
                            rmm::mr::device_memory_resource * mr)
{
    // TODO: Add check that edge sizes are > 0.
    CUDF_EXPECTS(input.type() == left_edges.type(), "The input and edge columns must have the same types.");
    CUDF_EXPECTS(input.type() == right_edges.type(), "The input and edge columns must have the same types.");
    CUDF_EXPECTS(left_edges.size() == right_edges.size(), "The left and right edge columns must be of the same length.");

    // Handle empty inputs.
    if (input.is_empty()) {
        // TODO: Determine what output type actually makes sense here, it
        // probably shouldn't be empty_like but instead of some numeric type.
        return empty_like(input);
    }

    return type_dispatcher(
            input.type(), bin_type_dispatcher{}, input, left_edges, left_inclusive, right_edges, right_inclusive, mr);
}
}  // namespace bin
}  // namespace cudf
