#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2021, NVIDIA CORPORATION.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <thrust/binary_search.h>
#include <thrust/execution_policy.h>
#include <thrust/functional.h>
#include <thrust/distance.h>
#include <thrust/advance.h>
#include <thrust/pair.h>
#include <cudf/binning/bin.hpp>
#include <cudf/column/column.hpp>
#include <cudf/column/column_device_view.cuh>
#include <cudf/column/column_factories.hpp>
#include <cudf/column/column_view.hpp>
#include <cudf/detail/valid_if.cuh>
#include <cudf/types.hpp>
#include <cudf/utilities/error.hpp>
#include <cudf/utilities/span.hpp>
#include <cudf/utilities/type_dispatcher.hpp>
#include <limits>
#include <rmm/mr/device/device_memory_resource.hpp>

namespace cudf {

// Sentinel used to indicate that an input value should be placed in the null
// bin.
// NOTE: In theory if a user decided to specify 2^31 bins this would fail. We
// could make this an error in Python, but that is such a crazy edge case...
constexpr size_type NULL_VALUE{std::numeric_limits<size_type>::max()};

namespace detail {
namespace {

template <typename T, typename RandomAccessIterator, typename LeftComparator, typename RightComparator>
struct bin_finder {
  bin_finder(RandomAccessIterator left_begin, RandomAccessIterator left_end, RandomAccessIterator right_begin, size_type edge_index_shift)
    : m_left_begin(left_begin), m_left_end(left_end), m_right_begin(right_begin), m_edge_index_shift(edge_index_shift)
  {
  }

  __device__ size_type operator()(thrust::pair<T, bool> input_value) const
  {
    // Immediately return sentinel for null inputs.
    if (!input_value.second) return NULL_VALUE;

    T value = input_value.first;
    auto bound =
      thrust::lower_bound(thrust::seq, m_left_begin, m_left_end, value, m_left_comp);

    // Exit early and return sentinel for values that lie below the interval.
    if (bound == m_left_begin) { return NULL_VALUE; }

    auto index = thrust::distance(m_left_begin, thrust::prev(bound));
    return (m_right_comp(value, m_right_begin[index])) ? (index + m_edge_index_shift) : NULL_VALUE;
  }

  RandomAccessIterator m_left_begin{};   // The beginning of the range containing the left bin edges.
  RandomAccessIterator m_left_end{};     // The end of the range containing the left bin edges.
  RandomAccessIterator m_right_begin{};  // The beginning of the range containing the right bin edges.
  size_type m_edge_index_shift;          // The number of elements m_left_begin has been shifted to skip nulls.
  LeftComparator m_left_comp{};          // Comparator used for left edges.
  RightComparator m_right_comp{};        // Comparator used for right edges.
};

// Functor to identify rows that should be filtered out based on the sentinel set by
// bin_finder::operator().
struct filter_null_sentinel {
  __device__ bool operator()(size_type i) { return i != NULL_VALUE; }
};

/// Bin the input by the edges in left_edges and right_edges.
template <typename T, typename LeftComparator, typename RightComparator>
std::unique_ptr<column> bin(column_view const& input,
                            column_view const& left_edges,
                            column_view const& right_edges,
                            null_order edge_null_precedence,
                            rmm::mr::device_memory_resource* mr)
{
  auto output = cudf::make_numeric_column(data_type(type_to_id<size_type>()), input.size());
  auto output_mutable_view = output->mutable_view();
  // These device column views are necessary for creating iterators that work
  // for columns of compound types. The column_view iterators do not work in
  // this case since they return raw pointers to the start of the data.
  auto input_device_view   = column_device_view::create(input);
  auto left_edges_device_view   = column_device_view::create(left_edges);
  auto right_edges_device_view   = column_device_view::create(right_edges);

  // Compute the maximum shift required for either edge, then shift all the iterators appropriately.
  size_type null_shift = max(left_edges.null_count(), right_edges.null_count());
  auto left_begin = left_edges_device_view->begin<T>();
  auto left_end = left_edges_device_view->end<T>();
  auto right_begin = right_edges_device_view->begin<T>();

  if (edge_null_precedence == null_order::BEFORE)
  {
      left_begin = thrust::next(left_begin, null_shift);
      right_begin = thrust::next(right_begin, null_shift);
  }
  else
  {
      left_end = thrust::prev(left_end, null_shift);
  }

  // If all the nulls are at the beginning, the indices found by lower_bound
  // will be off by null_shift, but if they're at the end the indices will
  // already be correct.
  size_type index_shift = (edge_null_precedence == null_order::BEFORE) ? null_shift : 0;

  if (input.has_nulls())
  {
      thrust::transform(thrust::device,
                        input_device_view->pair_begin<T, true>(),
                        input_device_view->pair_end<T, true>(),
                        output_mutable_view.begin<size_type>(),
                        bin_finder<T, decltype(left_edges_device_view->begin<T>()), LeftComparator, RightComparator>(
                          left_begin, left_end, right_begin, index_shift));
  }
  else
  {
      thrust::transform(thrust::device,
                        input_device_view->pair_begin<T, false>(),
                        input_device_view->pair_end<T, false>(),
                        output_mutable_view.begin<size_type>(),
                        bin_finder<T, decltype(left_edges_device_view->begin<T>()), LeftComparator, RightComparator>(
                          left_begin, left_end, right_begin, index_shift));
  }

  auto mask_and_count = cudf::detail::valid_if(output_mutable_view.begin<size_type>(),
                                               output_mutable_view.end<size_type>(),
                                               filter_null_sentinel());

  output->set_null_mask(mask_and_count.first, mask_and_count.second);
  return output;
}

template <typename T>
constexpr auto is_supported_bin_type()
{
  // TODO: Determine what other types (such as fixed point numbers) should be
  // supported, and whether any of them (like strings) require special
  // handling.
  return ((cudf::is_numeric<T>() && !std::is_same<T, bool>::value)) || std::is_same<T, cudf::string_view>::value;
}

}  // anonymous namespace
}  // namespace detail

/// Functor suitable for use with type_dispatcher that exploits SFINAE to call the appropriate
/// detail::bin method.
struct bin_type_dispatcher {
  template <typename T, typename... Args>
  std::enable_if_t<not detail::is_supported_bin_type<T>(), std::unique_ptr<column>> operator()(
    Args&&... args)
  {
    CUDF_FAIL("Type not support for cudf::bin");
  }

  template <typename T>
  std::enable_if_t<detail::is_supported_bin_type<T>(), std::unique_ptr<column>> operator()(
    column_view const& input,
    column_view const& left_edges,
    inclusive left_inclusive,
    column_view const& right_edges,
    inclusive right_inclusive,
    null_order edge_null_precedence,
    rmm::mr::device_memory_resource* mr)
  {
    // Using a switch statement might be more appropriate for an enum, but it's far more verbose
    // in this case.
    if ((left_inclusive == inclusive::YES) && (right_inclusive == inclusive::YES))
      return detail::bin<T, thrust::less_equal<T>, thrust::less_equal<T> >(
        input, left_edges, right_edges, edge_null_precedence, mr);
    if ((left_inclusive == inclusive::YES) && (right_inclusive == inclusive::NO))
      return detail::bin<T, thrust::less_equal<T>, thrust::less<T> >(
        input, left_edges, right_edges, edge_null_precedence, mr);
    if ((left_inclusive == inclusive::NO) && (right_inclusive == inclusive::YES))
      return detail::bin<T, thrust::less<T>, thrust::less_equal<T> >(
        input, left_edges, right_edges, edge_null_precedence, mr);
    if ((left_inclusive == inclusive::NO) && (right_inclusive == inclusive::NO))
      return detail::bin<T, thrust::less<T>, thrust::less<T> >(
        input, left_edges, right_edges, edge_null_precedence, mr);

    CUDF_FAIL("Undefined inclusive setting.");
  }
};

/// Bin the input by the edges in left_edges and right_edges.
std::unique_ptr<column> bin(column_view const& input,
                            column_view const& left_edges,
                            inclusive left_inclusive,
                            column_view const& right_edges,
                            inclusive right_inclusive,
                            null_order edge_null_precedence,
                            rmm::mr::device_memory_resource* mr)
{
  CUDF_FUNC_RANGE()
  CUDF_EXPECTS((input.type() == left_edges.type()) && (input.type() == right_edges.type()),
               "The input and edge columns must have the same types.");
  CUDF_EXPECTS(left_edges.size() == right_edges.size(),
               "The left and right edge columns must be of the same length.");

  // Handle empty inputs.
  if (input.is_empty()) { return cudf::make_numeric_column(data_type(type_to_id<size_type>()), 0); }

  return type_dispatcher<dispatch_storage_type>(input.type(),
                                                bin_type_dispatcher{},
                                                input,
                                                left_edges,
                                                left_inclusive,
                                                right_edges,
                                                right_inclusive,
                                                edge_null_precedence,
                                                mr);
}
}  // namespace cudf
